/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

#include "rtkCudaUtilities.hcu"

std::vector<int> GetListOfCudaDevices()
{
  std::vector<int>      deviceList;
  int                   deviceCount;
  struct hipDeviceProp_t properties;
  if (hipGetDeviceCount(&deviceCount) == hipSuccess)
    {
    for (int device = 0; device < deviceCount; ++device) {
      hipGetDeviceProperties(&properties, device);
      if (properties.major != 9999)   /* 9999 means emulation only */
        deviceList.push_back(device);
      }
    }
  if(deviceList.size()<1)
    itkGenericExceptionMacro(<< "No CUDA device available");

  return deviceList;
}

std::pair<int,int> GetCudaComputeCapability(int device)
{
  struct hipDeviceProp_t properties;
  if (hipGetDeviceProperties(&properties, device) != hipSuccess)
    itkGenericExceptionMacro(<< "Invalid CUDA device");
  return std::make_pair(properties.major, properties.minor);
}

size_t GetFreeGPUGlobalMemory(int device)
{
  // The return result of cuda utility methods are stored in a hipError_t
  hipError_t result;

  //create cuda context
  hipCtx_t cudaContext;
  result = hipCtxCreate(&cudaContext, hipDeviceScheduleAuto, device);
  if(result != hipSuccess)
    {
    itkGenericExceptionMacro(<< "Could not create context on this CUDA device");
    }

  //get the amount of free memory on the graphics card
  size_t free;
  size_t total;
  result = hipMemGetInfo(&free, &total);
  if(result != hipSuccess)
    {
    itkGenericExceptionMacro(<< "Could not obtain information on free memory on this CUDA device");
    }

  hipCtxDestroy(cudaContext);

  return free;
}
